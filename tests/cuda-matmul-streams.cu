#include "hip/hip_runtime.h"

#include <iostream>
#include <omp.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <string>

#define DEV
#define GLOBAL_MEMORY

constexpr int BLOCK_SIZE = 32;

using namespace std;

#define gpuErrCheck(ans)                      \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        cout << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << endl;
        if (abort)
        {
            exit(code);
        }
    }
}

void populateMatrixBuffer(float *buffer, int dimSize)
{
    for (int i = 0; i < dimSize; i++)
    {
        for (int j = 0; j < dimSize; j++)
        {
            buffer[i * dimSize + j] = 1.0f / (j + 1);
        }
    }
}

__global__ void matMulGPU(const float *matrixA, const float *matrixB, float *matrixC, int dimSize)
{
    const int x = blockIdx.x * BLOCK_SIZE + (threadIdx.x / BLOCK_SIZE);
    const int y = blockIdx.y * BLOCK_SIZE + (threadIdx.x % BLOCK_SIZE);

    if (x < dimSize && y < dimSize)
    {
        float product = 0.0f;
        for (int i = 0; i < dimSize; i++)
        {
            product += matrixA[x * dimSize + i] * matrixB[i * dimSize + y];
        }
        matrixC[x * dimSize + y] = product;
    }
}

int main()
{
    int MAX_DIM_SIZE = 16384;

    for (int DIM_SIZE = 64; DIM_SIZE <= MAX_DIM_SIZE; DIM_SIZE <<= 1)
    {
        std::cout << "-----------------------------------------------" << endl;
        std::cout << "DIM_SIZE: " << DIM_SIZE << endl
                  << endl;

        // Allocate host memory
        float *h_matrixA = new float[DIM_SIZE * DIM_SIZE];
        float *h_matrixB = new float[DIM_SIZE * DIM_SIZE];
        float *h_matrixC = new float[DIM_SIZE * DIM_SIZE];
        populateMatrixBuffer(h_matrixA, DIM_SIZE);
        populateMatrixBuffer(h_matrixB, DIM_SIZE);

        const float alpha = 1.0f;
        const float beta = 0.0f;

        float matMulCPUNaiveTimeMs = -1.0f;
        float matMulCPUIntelMKLTimeMs = -1.0f;
        float matMulGPUMs = -1.0f;
        float matMulGPUSharedMemMs = -1.0f;
        float matMulGPUcuBLASMs = -1.0f;

        // Create CUDA stream
        hipStream_t stream;
        gpuErrCheck(hipStreamCreate(&stream));

        // Allocate device memory asynchronously
        float *d_matrixA;
        float *d_matrixB;
        float *d_matrixC;
        gpuErrCheck(hipMallocAsync(&d_matrixA, DIM_SIZE * DIM_SIZE * sizeof(float), stream));
        gpuErrCheck(hipMallocAsync(&d_matrixB, DIM_SIZE * DIM_SIZE * sizeof(float), stream));
        gpuErrCheck(hipMallocAsync(&d_matrixC, DIM_SIZE * DIM_SIZE * sizeof(float), stream));

        // Copy data from host to device asynchronously
        gpuErrCheck(hipMemcpyAsync(d_matrixA, h_matrixA, DIM_SIZE * DIM_SIZE * sizeof(float), hipMemcpyHostToDevice, stream));
        gpuErrCheck(hipMemcpyAsync(d_matrixB, h_matrixB, DIM_SIZE * DIM_SIZE * sizeof(float), hipMemcpyHostToDevice, stream));

        // Init CUDA events used to meassure timings
        hipEvent_t startEvent, stopEvent;
        gpuErrCheck(hipEventCreate(&startEvent));
        gpuErrCheck(hipEventCreate(&stopEvent));

        // Define grids, blocks and threads
        const int GRID_SIZE = (DIM_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dim3 gridDim(GRID_SIZE, GRID_SIZE); // 2D Grid
        dim3 blockDim(BLOCK_SIZE * BLOCK_SIZE);

        {
            // Call kernel (global-memory)
            gpuErrCheck(hipEventRecord(startEvent, stream));
            matMulGPU<<<gridDim, blockDim>>>(d_matrixA, d_matrixB, d_matrixC, DIM_SIZE);
            gpuErrCheck(hipEventRecord(stopEvent, stream));
            gpuErrCheck(hipEventSynchronize(stopEvent));
            gpuErrCheck(hipEventElapsedTime(&matMulGPUMs, startEvent, stopEvent));
            gpuErrCheck(hipPeekAtLastError());
            std::cout << "GPU_GLOBAL time [ms]: " << matMulGPUMs << endl;
        }

        // Synchronize stream
        gpuErrCheck(hipStreamSynchronize(stream));

        // Free device memory asynchronously
        gpuErrCheck(hipFreeAsync(d_matrixA, stream));
        gpuErrCheck(hipFreeAsync(d_matrixB, stream));
        gpuErrCheck(hipFreeAsync(d_matrixC, stream));
        delete[] h_matrixA;
        delete[] h_matrixB;
        delete[] h_matrixC;
    }

    return 0;
}